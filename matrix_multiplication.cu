#include <iostream>
#include <hip/hip_runtime.h>

// CUDA kernel for matrix multiplication
__global__ void matrix_multiply(int* A, int* B, int* C, int width) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    

    if (row < width && col < width) {
        int sum = 0;
        for (int i = 0; i < width; i++) {
            sum += A[row * width + i] * B[i * width + col];
            printf("Block  x:%d,y:%d Thread x:%d,y:%d => Adding A[%d] + B[%d] to sum\n", blockIdx.x, blockIdx.y, threadIdx.x, threadIdx.y,row * width + i, i * width + col);
        }
        printf("Block  x:%d,y:%d Thread x:%d,y:%d => Saving sum to C[%d]\n", blockIdx.x, blockIdx.y, threadIdx.x, threadIdx.y,row * width + col);
        C[row * width + col] = sum;
    }
}

int main() {
    int width = 4;
    int size = width * width;

    int* h_A = new int[size];
    int* h_B = new int[size];
    int* h_C = new int[size];

    for (int i = 0; i < size; i++) {
        h_A[i] = i;
        h_B[i] = i;
    }

    int* d_A, * d_B, * d_C;
    hipMalloc(&d_A, size * sizeof(int));
    hipMalloc(&d_B, size * sizeof(int));
    hipMalloc(&d_C, size * sizeof(int));

    hipMemcpy(d_A, h_A, size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size * sizeof(int), hipMemcpyHostToDevice);

    dim3 block_dim(2, 2);
    dim3 grid_dim((width + block_dim.x - 1) / block_dim.x, (width + block_dim.y - 1) / block_dim.y);

    matrix_multiply<<<grid_dim, block_dim>>>(d_A, d_B, d_C, width);

    hipMemcpy(h_C, d_C, size * sizeof(int), hipMemcpyDeviceToHost);
    int m = 0;
    if (10 < width)
      m = 10;
    else
      m = width;
   
    printf("\nA\n");
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < m; j++) {
            std::cout << h_A[i * width + j] << " ";
        }
        std::cout << std::endl;
    }

    printf("\nB\n");
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < m; j++) {
            std::cout << h_B[i * width + j] << " ";
        }
        std::cout << std::endl;
    }

    printf("\nC\n");
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < m; j++) {
            std::cout << h_C[i * width + j] << " ";
        }
        std::cout << std::endl;
    }

    delete[] h_A;
    delete[] h_B;
    delete[] h_C;
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}
